#include "hip/hip_runtime.h"
#include <iostream>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

extern "C" {
#include "gemm.h"
}

#include <algorithm>
#include <cfloat>  // For FLT_MAX and FLT_MIN
#include <cmath>   // For std::abs

void cuda_sgemm_native(char* TRANS_A, char* TRANS_B,
                      int* M, int* N, int* K,
                      double* ALPHA,
                      double* dA, int* LDA,
                      double* dB, int* LDB,
                      double* BETA,
                      double* dC, int* LDC) {
    // Unpack
    int m = *M, n = *N, k = *K;
    char trans_a = *TRANS_A;
    char trans_b = *TRANS_B;
    double alpha = *ALPHA;
    double beta  = *BETA;
    int lda = *LDA, ldb = *LDB, ldc = *LDC;

    hipblasOperation_t TRANSA_symb = (trans_a == 'N') ? HIPBLAS_OP_N : HIPBLAS_OP_T;
    hipblasOperation_t TRANSB_symb = (trans_b == 'N') ? HIPBLAS_OP_N : HIPBLAS_OP_T;

    int a_rows = (trans_a == 'N') ? m : k;
    int a_cols = (trans_a == 'N') ? k : m;
    int b_rows = (trans_b == 'N') ? k : n;
    int b_cols = (trans_b == 'N') ? n : k;

    // Allocate host float arrays
    float* A = new float[a_rows * a_cols];
    float* B = new float[b_rows * b_cols];
    float* C = new float[m * n];

    auto clamp = [](double x) -> float {
        if (x >= FLT_MAX) return FLT_MAX;
        if (x <= -FLT_MAX) return -FLT_MAX;
        if (std::abs(x) > 0.0 && std::abs(x) < FLT_MIN)
            return (x > 0.0 ? FLT_MIN : -FLT_MIN);
        return static_cast<float>(x);
    };

    // Convert and copy A and B with clamp
    for (int col = 0; col < a_cols; ++col)
        for (int row = 0; row < a_rows; ++row)
            A[col * a_rows + row] = clamp(dA[col * lda + row]);

    for (int col = 0; col < b_cols; ++col)
        for (int row = 0; row < b_rows; ++row)
            B[col * b_rows + row] = clamp(dB[col * ldb + row]);

    // Convert C if beta != 0, otherwise zero initialize
    if (beta != 0.0) {
        for (int col = 0; col < n; ++col)
            for (int row = 0; row < m; ++row)
                C[col * m + row] = clamp(dC[col * ldc + row]);
    } else {
        std::fill(C, C + m * n, 0.0f);
    }

    // cuBLAS setup
    hipStream_t stream;
    hipblasHandle_t cublasH;
    hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
    hipblasCreate(&cublasH);
    hipblasSetStream(cublasH, stream);

    // Allocate device memory
    float *dat_A = nullptr, *dat_B = nullptr, *dat_C = nullptr;
    hipMallocAsync(&dat_A, sizeof(float) * a_rows * a_cols, stream);
    hipMallocAsync(&dat_B, sizeof(float) * b_rows * b_cols, stream);
    hipMallocAsync(&dat_C, sizeof(float) * m * n, stream);

    // Copy to device
    hipMemcpyAsync(dat_A, A, sizeof(float) * a_rows * a_cols, hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(dat_B, B, sizeof(float) * b_rows * b_cols, hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(dat_C, C, sizeof(float) * m * n, hipMemcpyHostToDevice, stream);

    float alpha_f = clamp(alpha);
    float beta_f = clamp(beta);

    // GEMM
    hipblasSgemm(cublasH,
                TRANSA_symb, TRANSB_symb,
                m, n, k,
                &alpha_f,
                dat_A, a_rows,
                dat_B, b_rows,
                &beta_f,
                dat_C, m);

    // Copy result back to host C
    hipMemcpyAsync(C, dat_C, sizeof(float) * m * n, hipMemcpyDeviceToHost, stream);

    // Wait
    hipStreamSynchronize(stream);

    // Write back to strided double dC with cast
    for (int col = 0; col < n; ++col)
        for (int row = 0; row < m; ++row)
            dC[col * ldc + row] = static_cast<double>(C[col * m + row]);

    // Cleanup
    hipFreeAsync(dat_A, stream);
    hipFreeAsync(dat_B, stream);
    hipFreeAsync(dat_C, stream);
    hipStreamSynchronize(stream);

    hipblasDestroy(cublasH);
    hipStreamDestroy(stream);
    delete[] A;
    delete[] B;
    delete[] C;
}
