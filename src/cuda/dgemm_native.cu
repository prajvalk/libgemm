#include <iostream>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

extern "C" {
#include "gemm.h"
}

void cuda_dgemm_native(char* TRANS_A, char* TRANS_B,
                      int* M, int* N, int* K,
                      double* ALPHA,
                      double* A, int* LDA,
                      double* B, int* LDB,
                      double* BETA,
                      double* C, int* LDC) {
    
    // Input parameters
    char trans_a = *TRANS_A;
    char trans_b = *TRANS_B;
    int m = *M;
    int n = *N;
    int k = *K;
    double alpha = *ALPHA;
    double beta = *BETA;
    int lda = *LDA;
    int ldb = *LDB;
    int ldc = *LDC;

    hipblasOperation_t TRANSA_symb = (trans_a == 'N') ? HIPBLAS_OP_N : HIPBLAS_OP_T;
    hipblasOperation_t TRANSB_symb = (trans_b == 'N') ? HIPBLAS_OP_N : HIPBLAS_OP_T;

    int a_rows = (trans_a == 'N') ? m : k;
    int a_cols = (trans_a == 'N') ? k : m;
    int b_rows = (trans_b == 'N') ? k : n;
    int b_cols = (trans_b == 'N') ? n : k;

    // Create stream and cuBLAS handle
    hipStream_t stream;
    hipblasHandle_t cublasH;
    hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
    hipblasCreate(&cublasH);
    hipblasSetStream(cublasH, stream);

    double *dat_A = NULL, *dat_B = NULL, *dat_C = NULL;

    // Async device allocations tied to stream
    hipMallocAsync(&dat_A, sizeof(double) * lda * a_cols, stream);
    hipMallocAsync(&dat_B, sizeof(double) * ldb * b_cols, stream);
    hipMallocAsync(&dat_C, sizeof(double) * ldc * n, stream);

    // Async copies on the same stream
    hipMemcpy2DAsync(dat_A, lda * sizeof(double), A, lda * sizeof(double),
                      a_rows * sizeof(double), a_cols, hipMemcpyHostToDevice, stream);

    hipMemcpy2DAsync(dat_B, ldb * sizeof(double), B, ldb * sizeof(double),
                      b_rows * sizeof(double), b_cols, hipMemcpyHostToDevice, stream);

    hipMemcpy2DAsync(dat_C, ldc * sizeof(double), C, ldc * sizeof(double),
                      m * sizeof(double), n, hipMemcpyHostToDevice, stream);

    // Queue GEMM after copies
    hipblasDgemm(cublasH,
                TRANSA_symb, TRANSB_symb,
                m, n, k,
                &alpha,
                dat_A, lda,
                dat_B, ldb,
                &beta,
                dat_C, ldc);

    // Async copy result back
    hipMemcpy2DAsync(C, ldc * sizeof(double), dat_C, ldc * sizeof(double),
                      m * sizeof(double), n, hipMemcpyDeviceToHost, stream);

    // Wait for all GPU work on stream to complete
    hipStreamSynchronize(stream);

    // Async free device memory
    hipFreeAsync(dat_A, stream);
    hipFreeAsync(dat_B, stream);
    hipFreeAsync(dat_C, stream);

    // Clean up handles and stream
    hipblasDestroy(cublasH);
    hipStreamDestroy(stream);
}