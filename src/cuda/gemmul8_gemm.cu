#include "hip/hip_runtime.h"
#include <iostream>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

#include <cmath>    
#include <cfloat>  
#include <algorithm> 

extern "C" {
#include "gemm.h"
}

#include "gemmul8/gemmul8.hpp"

#ifdef GEMM8_DEBUG_MODE
#include "matrix.hpp"
#include "matrixmarketio.hpp"
#endif

void sanitize_matrix(double* C, int rows, int cols, int ldc) {
    for (int col = 0; col < cols; ++col) {
        for (int row = 0; row < rows; ++row) {
            double& val = C[col * ldc + row];
            if (std::isnan(val)) {
                val = 0.0; // or some other neutral/default value
            } else if (std::isinf(val)) {
                val = (val > 0) ? DBL_MAX : -DBL_MAX;
            } else {
                // Optional: clamp to finite range if needed
                val = std::min(std::max(val, -DBL_MAX), DBL_MAX);
            }
        }
    }
}

void gemmul8_gemm (char* TRANSA, 
             char* TRANSB, 
             int*  M, 
             int*  N, 
             int*  K, 
             double* ALPHA,
	         double* A, int* LDA, 
             double* B, int* LDB, 
             double* BETA, 
             double* C, int* LDC,
             int moduli, bool fastmode) {
    // Input parameters
    char trans_a = *TRANSA;
    char trans_b = *TRANSB;
    int m = *M;
    int n = *N;
    int k = *K;
    double alpha = *ALPHA;
    double beta = *BETA;
    int lda = *LDA;
    int ldb = *LDB;
    int ldc = *LDC;
    int LWORK = gemmul8::workSize(m, n, k, moduli);

    hipblasOperation_t TRANSA_symb = (trans_a == 'N') ? HIPBLAS_OP_N : HIPBLAS_OP_T;
    hipblasOperation_t TRANSB_symb = (trans_b == 'N') ? HIPBLAS_OP_N : HIPBLAS_OP_T;

    int a_rows = (trans_a == 'N') ? m : k;
    int a_cols = (trans_a == 'N') ? k : m;
    int b_rows = (trans_b == 'N') ? k : n;
    int b_cols = (trans_b == 'N') ? n : k;
    int c_rows = m;
    int c_cols = n;

    if (lda < std::max(1, a_rows)) {
        std::cerr << "Invalid lda: " << lda << " < " << std::max(1, a_rows) << "\n";
    }

    if (ldb < std::max(1, b_rows)) {
        std::cerr << "Invalid ldb: " << ldb << " < " << std::max(1, b_rows) << "\n";
    }

    if (ldc < std::max(1, c_rows)) {
        std::cerr << "Invalid ldc: " << ldc << " < " << std::max(1, c_rows) << "\n";
    }

    // Create stream and cuBLAS handle
    hipStream_t stream;
    hipblasHandle_t cublasH;
    hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
    hipblasCreate(&cublasH);
    hipblasSetStream(cublasH, stream);

    double *dat_A = NULL, *dat_B = NULL, *dat_C = NULL;
    void *WORK = NULL;
#ifdef GEMM8_DEBUG_MODE
    double *dat_Cref;
    double *Cref = new double[ldc * n];
#endif

    // Async device allocations tied to stream
    hipMallocAsync(&dat_A, sizeof(double) * lda * a_cols, stream);
    hipMallocAsync(&dat_B, sizeof(double) * ldb * b_cols, stream);
    hipMallocAsync(&dat_C, sizeof(double) * ldc * n, stream);
#ifdef GEMM8_DEBUG_MODE
    hipMallocAsync(&dat_Cref, sizeof(double) * ldc * n, stream);
#endif
    hipMallocAsync(&WORK, LWORK, stream);
    hipMemsetAsync(WORK, 0, LWORK, stream);

    // Async copies on the same stream
    hipMemcpy2DAsync(dat_A, lda * sizeof(double), A, lda * sizeof(double),
                      a_rows * sizeof(double), a_cols, hipMemcpyHostToDevice, stream);

    hipMemcpy2DAsync(dat_B, ldb * sizeof(double), B, ldb * sizeof(double),
                      b_rows * sizeof(double), b_cols, hipMemcpyHostToDevice, stream);

    hipMemcpy2DAsync(dat_C, ldc * sizeof(double), C, ldc * sizeof(double),
                      m * sizeof(double), n, hipMemcpyHostToDevice, stream);
#ifdef GEMM8_DEBUG_MODE
    hipMemcpy2DAsync(dat_Cref, ldc * sizeof(double), C, ldc * sizeof(double),
                      m * sizeof(double), n, hipMemcpyHostToDevice, stream);
#endif

    hipStreamSynchronize(stream);

    // Queue GEMM after copies
#ifdef GEMM8_DEBUG_MODE
    hipblasDgemm(cublasH,
                TRANSA_symb, TRANSB_symb,
                m, n, k,
                &alpha,
                dat_A, lda,
                dat_B, ldb,
                &beta,
                dat_Cref, ldc);
#endif

    gemmul8::gemm(cublasH,
                TRANSA_symb, TRANSB_symb,
                m, n, k,
                &alpha,
                dat_A, lda,
                dat_B, ldb,
                &beta,
                dat_C, ldc, moduli, fastmode, WORK);

    hipStreamSynchronize(stream);

#ifdef GEMM8_DEBUG_MODE
    // Async copy result back
    hipMemcpy2DAsync(Cref, ldc * sizeof(double), dat_Cref, ldc * sizeof(double),
                      m * sizeof(double), n, hipMemcpyDeviceToHost, stream);
#endif

    hipMemcpy2DAsync(C, ldc * sizeof(double), dat_C, ldc * sizeof(double),
                      m * sizeof(double), n, hipMemcpyDeviceToHost, stream);

    // Wait for all GPU work on stream to complete
    hipStreamSynchronize(stream);

#ifdef GEMM8_DEBUG_MODE
for (int cols = 0; cols < c_cols; ++cols) {
    for (int rows = 0; rows < c_rows; ++rows) {
        double diff = Cref[cols * ldc + rows] - C[cols * ldc + rows];
        if (std::abs(diff) > 1e-6) {
            printf("Mismatch at [%d,%d]: ref=%.12f custom=%.12f diff=%.12f\n",
                   rows, cols,
                   Cref[cols * ldc + rows],
                   C[cols * ldc + rows],
                   diff);
        }
    }
}
#endif

#ifdef GEMM8_DEBUG_MODE
    double l2 = 0;

    for (int cols = 0; cols < c_cols; cols++)
        for (int rows  = 0; rows < c_rows; rows++)
            l2 += pow(Cref[cols * ldc + rows] - C[cols * ldc + rows], 2);

    l2 = sqrt(l2);
    
    printf("L2: %f \n", l2);
#endif

    // Async free device memory
    hipFreeAsync(dat_A, stream);
    hipFreeAsync(dat_B, stream);
    hipFreeAsync(dat_C, stream);
    hipFreeAsync(WORK, stream);

#ifdef GEMM8_DEBUG_MODE
    hipFreeAsync(dat_Cref, stream);
    delete[] Cref;
#endif

    // Clean up handles and stream
    hipblasDestroy(cublasH);
    hipStreamDestroy(stream);
}

/*void gemmul8_gemm_new_old (char* TRANSA, 
             char* TRANSB, 
             int*  M, 
             int*  N, 
             int*  K, 
             double* ALPHA,
	         double* A, int* LDA, 
             double* B, int* LDB, 
             double* BETA, 
             double* C, int* LDC,
             int moduli, bool fastmode) 
{
    // Input parameters
    char trans_a = *TRANSA;
    char trans_b = *TRANSB;
    int m = *M;
    int n = *N;
    int k = *K;
    double alpha = *ALPHA;
    double beta = *BETA;
    int lda = *LDA;
    int ldb = *LDB;
    int ldc = *LDC;
    int LWORK = gemmul8::workSize(m, n, k, moduli);

    hipblasOperation_t TRANSA_symb = (trans_a == 'N') ? HIPBLAS_OP_N : HIPBLAS_OP_T;
    hipblasOperation_t TRANSB_symb = (trans_b == 'N') ? HIPBLAS_OP_N : HIPBLAS_OP_T;

    int a_rows = (trans_a == 'N') ? m : k;
    int a_cols = (trans_a == 'N') ? k : m;
    int b_rows = (trans_b == 'N') ? k : n;
    int b_cols = (trans_b == 'N') ? n : k;

    // Create stream and cuBLAS handle
    hipStream_t stream;
    hipblasHandle_t cublasH;
    hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
    hipblasCreate(&cublasH);
    hipblasSetStream(cublasH, stream);

    double *dat_A = NULL, *dat_B = NULL, *dat_C = NULL;
    void *WORK = NULL;

    // Async device allocations tied to stream
    hipMallocAsync(&dat_A, sizeof(double) * lda * a_cols, stream);
    hipMallocAsync(&dat_B, sizeof(double) * ldb * b_cols, stream);
    hipMallocAsync(&dat_C, sizeof(double) * ldc * n, stream);
    hipMallocAsync(&WORK, LWORK, stream);

    // Async copies on the same stream
    hipMemcpy2DAsync(dat_A, lda * sizeof(double), A, lda * sizeof(double),
                      a_rows * sizeof(double), a_cols, hipMemcpyHostToDevice, stream);

    hipMemcpy2DAsync(dat_B, ldb * sizeof(double), B, ldb * sizeof(double),
                      b_rows * sizeof(double), b_cols, hipMemcpyHostToDevice, stream);

    hipMemcpy2DAsync(dat_C, ldc * sizeof(double), C, ldc * sizeof(double),
                      m * sizeof(double), n, hipMemcpyHostToDevice, stream);

    hipMemsetAsync(WORK, 0, LWORK, stream);

    hipStreamSynchronize(stream);

    // Queue GEMMul8 after copies
    gemmul8::gemm (cublasH,
                TRANSA_symb, TRANSB_symb,
                m, n, k,
                &alpha,
                dat_A, lda,
                dat_B, ldb,
                &beta,
                dat_C, ldc, 
                moduli, fastmode, WORK);

    hipDeviceSynchronize();

    // Async copy result back
    hipMemcpy2DAsync(C, ldc * sizeof(double), dat_C, ldc * sizeof(double),
                      m * sizeof(double), n, hipMemcpyDeviceToHost, stream);

    bool flag  = false;

#ifdef GEMM8_DEBUG_MODE
    double *dat_Cref = NULL;
    hipMallocAsync(&dat_C, sizeof(double) * ldc * n, stream);

    hipblasDgemm(cublasH,
            TRANSA_symb, TRANSB_symb,
            m, n, k,
            &alpha,
            dat_A, lda,
            dat_B, ldb,
            &beta,
            dat_Cref, ldc);

    double *Cref = new double[ldc * n];

    hipMemcpy2DAsync(Cref, ldc * sizeof(double), dat_Cref, ldc * sizeof(double),
                      m * sizeof(double), n, hipMemcpyDeviceToHost, stream);

    hipStreamSynchronize(stream);
    
    double l2_err = 0;
    for (int col = 0; col < n; ++col)
        for (int row = 0; row < m; ++row)
            l2_err += pow(C[col * ldc + row] - Cref[col * ldc + row], 2);

    l2_err = sqrt(l2_err);

    if(l2_err >= 1e-3) {
        printf("L2: %d \n", l2_err);

        printf("%c %c %d %d %d", trans_a, trans_b, lda, ldb, ldc);

        Matrix<double> matA (a_rows, a_cols);
        Matrix<double> matB (b_rows, b_cols);

        for (int col = 0; col < a_cols; ++col)
            for (int row = 0; row < a_rows; ++row)
                matA.set(row, col, A[lda * col + row]);

        for (int col = 0; col < b_cols; ++col)
            for (int row = 0; row < b_rows; ++row)
                matB.set(row, col, B[ldb * col + row]);

        save_matrix("os2_debug_A.mat", COORDINATE, matA);
        save_matrix("os2_debug_B.mat", COORDINATE, matB);

        flag = true;
    }
    
    hipFreeAsync(dat_Cref, stream);
    delete[] Cref;
#endif

    // Sanitization step
    sanitize_matrix(C, m, n, ldc);

    // Async free device memory
    hipFreeAsync(dat_A, stream);
    hipFreeAsync(dat_B, stream);
    hipFreeAsync(dat_C, stream);
    hipFreeAsync(WORK, stream);

    // Wait for all GPU work on stream to complete
    hipStreamSynchronize(stream);

    // Clean up handles and stream
    hipblasDestroy(cublasH);
    hipStreamDestroy(stream);

    if (flag) exit (-1);
}*/

/*void gemmul8_gemm_old (char* TRANSA, 
             char* TRANSB, 
             int*  M, 
             int*  N, 
             int*  K, 
             double* ALPHA,
	         double* A, int* LDA, 
             double* B, int* LDB, 
             double* BETA, 
             double* C, int* LDC,
             int moduli, bool fastmode) 
{
    // Input parameters
    char trans_a = *TRANSA;
    char trans_b = *TRANSB;
    int m = *M;
    int n = *N;
    int k = *K;
    double alpha = *ALPHA;
    double beta = *BETA;
    int lda = *LDA;
    int ldb = *LDB;
    int ldc = *LDC;

    int LWORK = gemmul8::workSize(m, n, k, moduli);

    hipblasOperation_t TRANSA_symb = (trans_a == 'N') ? HIPBLAS_OP_N : HIPBLAS_OP_T;
    hipblasOperation_t TRANSB_symb = (trans_b == 'N') ? HIPBLAS_OP_N : HIPBLAS_OP_T;

    int a_rows = (trans_a == 'N') ? m : k;
    int a_cols = (trans_a == 'N') ? k : m;
    int b_rows = (trans_b == 'N') ? k : n;
    int b_cols = (trans_b == 'N') ? n : k;

    // Allocate tightly packed host arrays
    double* A_pack = new double[a_rows * a_cols];
    double* B_pack = new double[b_rows * b_cols];
    double* C_pack = new double[m * n];

    // Pack A
    for (int col = 0; col < a_cols; ++col)
        for (int row = 0; row < a_rows; ++row)
            A_pack[col * a_rows + row] = A[col * lda + row];

    // Pack B
    for (int col = 0; col < b_cols; ++col)
        for (int row = 0; row < b_rows; ++row)
            B_pack[col * b_rows + row] = B[col * ldb + row];

    // Pack C if needed
    //if (beta != 0.0) {
        for (int col = 0; col < n; ++col)
            for (int row = 0; row < m; ++row)
                C_pack[col * m + row] = C[col * ldc + row];
    //}

    // Create stream and cuBLAS handle
    hipStream_t stream;
    hipblasHandle_t cublasH;
    hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
    hipblasCreate(&cublasH);
    hipblasSetStream(cublasH, stream);

    double *dat_A = NULL, *dat_B = NULL, *dat_C = NULL;
    void *WORK = NULL;

    hipError_t err = hipSuccess;

    // Allocate device memory for packed buffers
    err = hipMallocAsync(&dat_A, sizeof(double) * a_rows * a_cols, stream);
    //if(err != hipSuccess) printf("malloc(A) error");

    err = hipMallocAsync(&dat_B, sizeof(double) * b_rows * b_cols, stream);
    //if(err != hipSuccess) printf("malloc(B) error");

    err = hipMallocAsync(&dat_C, sizeof(double) * m * n, stream);
    //if(err != hipSuccess) printf("malloc(C) error");

    err = hipMallocAsync(&WORK, LWORK, stream);
    //if(err != hipSuccess) printf("malloc(WORK) error");

    err = hipMemsetAsync(WORK, 0, LWORK, stream);
    //if(err != hipSuccess) printf("memset(WORK) error");

    // Copy packed data to device
    hipMemcpyAsync(dat_A, A_pack, sizeof(double) * a_rows * a_cols, hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(dat_B, B_pack, sizeof(double) * b_rows * b_cols, hipMemcpyHostToDevice, stream);
    //if (beta != 0.0)
        hipMemcpyAsync(dat_C, C_pack, sizeof(double) * m * n, hipMemcpyHostToDevice, stream);

    hipStreamSynchronize(stream);

    // GEMMul8 compute
    gemmul8::gemm (cublasH,
                TRANSA_symb, TRANSB_symb,
                m, n, k,
                &alpha,
                dat_A, a_rows,
                dat_B, b_rows,
                &beta,
                dat_C, m,
                moduli, fastmode, WORK);

    //printf("finished");

    // Copy result back to host
    hipMemcpyAsync(C_pack, dat_C, sizeof(double) * m * n, hipMemcpyDeviceToHost, stream);

    // Sync before unpacking result
    hipStreamSynchronize(stream);

    // Unpack C to strided layout
    for (int col = 0; col < n; ++col)
        for (int row = 0; row < m; ++row)
            C[col * ldc + row] = C_pack[col * m + row];

    // Cleanup
    delete[] A_pack;
    delete[] B_pack;
    delete[] C_pack;

    hipFreeAsync(dat_A, stream);
    hipFreeAsync(dat_B, stream);
    hipFreeAsync(dat_C, stream);
    hipFreeAsync(WORK, stream);

    hipblasDestroy(cublasH);
    hipStreamDestroy(stream);
}*/